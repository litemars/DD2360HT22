

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <stdlib.h>

#define DataType double


__global__ void vecAdd(DataType *in1, DataType *in2, DataType *out, int len) {
  //@@ Insert code to implement vector addition here
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    
    if(id < len) {
      out[id] = in1[id] + in2[id];
      }
}

//@@ Insert code to implement timer start
double startTimer() {
   struct timeval tp;
   gettimeofday(&tp,NULL);
   return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}
//@@ Insert code to implement timer stop
double stopTimer(double startime) {
   struct timeval tp;
   gettimeofday(&tp,NULL);
   return (((double)tp.tv_sec + (double)tp.tv_usec*1.e-6) - startime);
}

double RandomReal(double low, double high)
{
  double d;

  d = (double) rand() / ((double) RAND_MAX + 1);
  return (low + d * (high - low));
}

void printArray(double *in, int len){
  for(int i=0;i<len;i++){
    printf("%f ",in[i]);
  }
  printf("\n\n");
}

int main(int argc, char **argv) {
  
  int inputLength;
  DataType *hostInput1;
  DataType *hostInput2;
  DataType *hostOutput;
  DataType *resultRef;
  DataType *deviceInput1;
  DataType *deviceInput2;
  DataType *deviceOutput;

  //@@ Insert code below to read in inputLength from args
  if (argc<2){
    printf("input length invalid\n");
    return 0;
  }
  inputLength=atoi(argv[1]);
  printf("The input length is %d\n", inputLength);
  
  //@@ Insert code below to allocate Host memory for input and output
  hostInput1 = (DataType*)malloc(inputLength*sizeof(DataType));
  hostInput2 = (DataType*)malloc(inputLength*sizeof(DataType));
  hostOutput = (DataType*)malloc(inputLength*sizeof(DataType));
  
  //@@ Insert code below to initialize hostInput1 and hostInput2 to random numbers, and create reference result in CPU

  for(int i=0;i<inputLength;i++){
      hostInput1[i]=RandomReal(0,1);
      hostInput2[i]=RandomReal(0,1);
  }

  resultRef = (DataType*)malloc(inputLength*sizeof(DataType));
  double start_time=startTimer();
  for(int i=0;i<inputLength;i++){
    resultRef[i]=hostInput1[i]+hostInput2[i];
  }
  double stopCPU=stopTimer(start_time);
  //@@ Insert code below to allocate GPU memory here
  hipMalloc(&deviceInput1, inputLength*sizeof(DataType));
  hipMalloc(&deviceInput2, inputLength*sizeof(DataType));
  hipMalloc(&deviceOutput, inputLength*sizeof(DataType));

  //@@ Insert code to below to Copy memory to the GPU here
  double start_mem=startTimer();
  hipMemcpy(deviceInput1, hostInput1, inputLength*sizeof(DataType), hipMemcpyHostToDevice);
  hipMemcpy(deviceInput2, hostInput2, inputLength*sizeof(DataType), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  double stop_mem=stopTimer(start_mem);
  //@@ Initialize the 1D grid and block dimensions here
  dim3 Dg(1024,1,1);
  dim3 Db(1024,1,1);

  //@@ Launch the GPU Kernel here
  double start_time_gpu=startTimer();
  vecAdd<<<Dg,Db>>>(deviceInput1,deviceInput2,deviceOutput,inputLength);
  hipDeviceSynchronize();
  double stopGPU=stopTimer(start_time_gpu);

  double start_mem_2=startTimer();
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostOutput, deviceOutput, inputLength*sizeof(DataType), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  double stop_mem_2=stopTimer(start_mem_2);

  //@@ Insert code below to compare the output with the reference
  for(int i=0;i<inputLength;i++){
    if(resultRef[i] != hostOutput[i] && abs(resultRef[i]-hostOutput[i])>0.001 ){
        printf("Error counting numbers: %f",abs(resultRef[i]-hostOutput[i]) );
        return 0;
    }
  }
  printf("sum verified: Correct!\n");
  printf("Time Host->Device: %f - Time Device->Host: %f\n",stop_mem,stop_mem_2);
  printf("CPU time: %f - GPU time: %f\n",stopCPU,stopGPU);
  
  //@@ Free the GPU memory here
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);


  //@@ Free the CPU memory here

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);

  return 0;
}
